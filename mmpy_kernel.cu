#include "hip/hip_runtime.h"
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include "utils.h"
#include "types.h"
using namespace std;
#define TW 32
#include <stdio.h>
__global__ void matMul(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {
	//if(N%TW){

	//	int I =  blockIdx.y*blockDim.y + threadIdx.y;
	//	int J =  blockIdx.x*blockDim.x + threadIdx.x;

	//	if((I < N) && (J < N)){
	//		_DOUBLE_ _c = 0;
	//		for (unsigned int k = 0; k < N; k++) {
	//			_DOUBLE_ a = A[I * N + k];
	//			_DOUBLE_ b = B[k * N + J];
	//			_c += a * b;
	//		}
	//		C[I * N + J] = _c;
	//	}
	//}
	//else
	//{
	__shared__ double As[TW][TW], Bs[TW][TW];
	int ty = threadIdx.y, tx = threadIdx.x;
	int by = blockIdx.y, bx = blockIdx.x;
	int I = min(N-1,by*TW + ty); 
	double Cij = 0;
	int J= min(N-1,bx*TW + tx);
	for (int kk=0; kk<ceilf(float (N)/TW); kk++)
	{
		As[ty][tx] = A[I*N + kk*TW+tx];
		Bs[ty][tx] = B[(kk*TW+ty)*N + J];
		__syncthreads();
		//printf("Loaded A[index=%d]<-%f,B[index=%d]<-%f\t:\tbx=%d,by=%d,tx=%d,ty=%d,As=%f,Bs=%f\n",I*N + kk*TW+tx,A[I*N + kk*TW+tx],(kk*TW+ty)*N + J,B[(kk*TW+ty)*N + J],bx,by,tx,ty,As[ty][tx],Bs[ty][tx]);
	//	if(I<N && J<N)
	//	{
			for (int k=0; k<TW && k+kk*TW<N; k++)
			{
				Cij+= As[ty][k] * Bs[k][tx];
		//		printf("N=%d, I=%d,J=%d, bx=%d,by=%d, tx=%d,ty=%d,kk=%d,k=%d,As=%f,index=%d,Aaddr=%p,Bs=%f,index=%d,Baddr=%p,Cij=%f\n",N,I,J,bx,by,tx,ty,kk,k,As[ty][k],I*N+kk*TW+k,(void *)&As[ty][k],Bs[k][tx],(kk*TW+k)*N+J,(void *)&Bs[k][tx],Cij);
			}
	//	}
		__syncthreads();
	}
//	if(I<N && J<N)
//	{
		C[I*N + J] = Cij;
	//	printf("Storing Cij=%f to C[%d] as %f at %p\n",Cij,I*N+J,C[I*N+J],(void *)&C[I*N+J]);
//	}
	//}
}
