#include "hip/hip_runtime.h"
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include "utils.h"
#include "types.h"
using namespace std;
#define TW 32
//#define TWx  32
//#define TWy  8
#include <stdio.h>
__global__ void matMul(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {
	__shared__ double As[TW][TW], Bs[TW][TW];
	int ty = threadIdx.y, tx = threadIdx.x;
	int by = blockIdx.y, bx = blockIdx.x;
	double Cij = 0;
	double Cij_4 = 0;
	double Cij_8 = 0;
	double Cij_12 = 0;
	double Cij_16 = 0;
	double Cij_20 = 0;
	double Cij_24 = 0;
	double Cij_28 = 0;
	if(N%TW)
{
	//ty = min(7,ty);
	int I = min(N-1,by*TW + ty); 
	int J= min(N-1,bx*TW + tx);

	
//	for (int kk=0; kk<ceilf(float (N)/TW); kk++)
	#pragma unroll
	for (int kk=0; kk<(N/TW + int(bool(N%TW))); kk++)
	{
		As[ty][tx] = A[I*N + kk*TW + tx];
		Bs[ty][tx] = B[(kk*TW+ty)*N + J];
		As[ty+4][tx] = A[(I+4)*N + kk*TW + tx];
		Bs[ty+4][tx] = B[(kk*TW+ty+4)*N + J];
		//printf("As = %f A = %f\n",As[ty][tx],A[I*N+kk*TWx+tx]); 
		//printf("ty = %d\n",ty);
		As[ty+8][tx] = A[(I+8)*N + kk*TW+tx];
		Bs[ty+8][tx] = B[(kk*TW+ty+8)*N + J];
		As[ty+12][tx] = A[(I+12)*N + kk*TW+tx];
		Bs[ty+12][tx] = B[(kk*TW+ty+12)*N + J];
		//printf("ty_8\n");
		As[ty+16][tx] = A[(I+16)*N + kk*TW+tx];
		Bs[ty+16][tx] = B[(kk*TW+ty+16)*N + J];
		As[ty+20][tx] = A[(I+20)*N + kk*TW+tx];
		Bs[ty+20][tx] = B[(kk*TW+ty+20)*N + J];
		//printf("ty_16\n");
		As[ty+24][tx] = A[(I+24)*N + kk*TW+tx];
		Bs[ty+24][tx] = B[(kk*TW+ty+24)*N + J];
		As[ty+28][tx] = A[(I+28)*N + kk*TW+tx];
		Bs[ty+28][tx] = B[(kk*TW+ty+28)*N + J];
		//printf("ty_24\n");
		__syncthreads();
			//for (int k=0; k<TW && k+kk*TW<N; k++)
			#pragma unroll
			for (int k=0; k<min(TW,N-kk*TW); k++)
			{
				Cij+= As[ty][k] * Bs[k][tx];
				Cij_4+= As[ty+4][k] * Bs[k][tx];
				Cij_8+= As[ty+8][k] * Bs[k][tx];
				Cij_12+= As[ty+12][k] * Bs[k][tx];
				Cij_16+= As[ty+16][k] * Bs[k][tx];
				Cij_20+= As[ty+20][k] * Bs[k][tx];
				Cij_24+= As[ty+24][k] * Bs[k][tx];
				Cij_28+= As[ty+28][k] * Bs[k][tx];
			}
		__syncthreads();
	}
		C[I*N + J] = Cij;
		C[(I+4)*N + J] = Cij_4;
		C[(I+8)*N + J] = Cij_8;
		C[(I+12)*N + J] = Cij_12;
		C[(I+16)*N + J] = Cij_16;
		C[(I+20)*N + J] = Cij_20;
		C[(I+24)*N + J] = Cij_24;
		C[(I+28)*N + J] = Cij_28;
}
else
{
	int I = by*TW + ty; 
	int J = bx*TW + tx;
//	for (int kk=0; kk<ceilf(float (N)/TW); kk++)
	if((I < N) && (J < N)) { 
	#pragma unroll
	for (int kk=0; kk<N/TW; kk++)
	{
		As[ty][tx] = A[I*N + kk*TW + tx];
		Bs[ty][tx] = B[(kk*TW+ty)*N + J];
		As[ty+4][tx] = A[(I+4)*N + kk*TW + tx];
		Bs[ty+4][tx] = B[(kk*TW+ty+4)*N + J];
		//printf("As = %f A = %f\n",As[ty][tx],A[I*N+kk*TWx+tx]); 
		//printf("ty = %d\n",ty);
		As[ty+8][tx] = A[(I+8)*N + kk*TW+tx];
		Bs[ty+8][tx] = B[(kk*TW+ty+8)*N + J];
		As[ty+12][tx] = A[(I+12)*N + kk*TW+tx];
		Bs[ty+12][tx] = B[(kk*TW+ty+12)*N + J];
		//printf("ty_8\n");
		As[ty+16][tx] = A[(I+16)*N + kk*TW+tx];
		Bs[ty+16][tx] = B[(kk*TW+ty+16)*N + J];
		As[ty+20][tx] = A[(I+20)*N + kk*TW+tx];
		Bs[ty+20][tx] = B[(kk*TW+ty+20)*N + J];
		//printf("ty_16\n");
		As[ty+24][tx] = A[(I+24)*N + kk*TW+tx];
		Bs[ty+24][tx] = B[(kk*TW+ty+24)*N + J];
		As[ty+28][tx] = A[(I+28)*N + kk*TW+tx];
		Bs[ty+28][tx] = B[(kk*TW+ty+28)*N + J];
		__syncthreads();
			//for (int k=0; k<TW && k+kk*TW<N; k++)
			#pragma unroll
			for (int k=0; k<TW; k++)
			{
				Cij+= As[ty][k] * Bs[k][tx];
				Cij_4+= As[ty+4][k] * Bs[k][tx];
				Cij_8+= As[ty+8][k] * Bs[k][tx];
				Cij_12+= As[ty+12][k] * Bs[k][tx];
				Cij_16+= As[ty+16][k] * Bs[k][tx];
				Cij_20+= As[ty+20][k] * Bs[k][tx];
				Cij_24+= As[ty+24][k] * Bs[k][tx];
				Cij_28+= As[ty+28][k] * Bs[k][tx];
			}
		__syncthreads();
		}
		C[I*N + J] = Cij;
		C[(I+4)*N + J] = Cij_4;
		C[(I+8)*N + J] = Cij_8;
		C[(I+12)*N + J] = Cij_12;
		C[(I+16)*N + J] = Cij_16;
		C[(I+20)*N + J] = Cij_20;
		C[(I+24)*N + J] = Cij_24;
		C[(I+28)*N + J] = Cij_28;
		}
	}
}
