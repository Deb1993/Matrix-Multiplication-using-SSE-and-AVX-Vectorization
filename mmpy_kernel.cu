#include "hip/hip_runtime.h"
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include "utils.h"
#include "types.h"
using namespace std;
#define TW BLOCKDIM_X//32
//#define TWx  32
//#define TWy  8
#include <stdio.h>
__global__ void matMul(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {
	__shared__ double As[TW][TW], Bs[TW][TW];
	int ty = threadIdx.y, tx = threadIdx.x;
	int by = blockIdx.y, bx = blockIdx.x;
	double Cij = 0;
	double Cij_4 = 0;
	double Cij_8 = 0;
	double Cij_12 = 0;
	double Cij_16 = 0;
	double Cij_20 = 0;
	double Cij_24 = 0;
	double Cij_28 = 0;
	if(N%TW || BLOCKDIM_X != BLOCKDIM_Y*8)
{
	//ty = min(7,ty);
	int I = min(N-1,by*TW + ty); 
	int J= min(N-1,bx*TW + tx);

	
//	for (int kk=0; kk<ceilf(float (N)/TW); kk++)
	if((I < N) && (J < N)) {
	#pragma unroll
	for (int kk=0; kk<(N/TW + int(bool(N%TW))); kk++)
	{
		As[ty][tx] = __ldg(&A[I*N + kk*TW + tx]);
		Bs[ty][tx] = __ldg(&B[(kk*TW+ty)*N + J]);
		//As[ty+4][tx] = __ldg(&A[(I+4)*N + kk*TW + tx]);
		//Bs[ty+4][tx] = __ldg(&B[(kk*TW+ty+4)*N + J]);
		////printf("As = %f A = %f\n",As[ty][tx],A[I*N+kk*TWx+tx]); 
		////printf("ty = %d\n",ty);
		//As[ty+8][tx] = __ldg(&A[(I+8)*N + kk*TW+tx]);
		//Bs[ty+8][tx] = __ldg(&B[(kk*TW+ty+8)*N + J]);
		//As[ty+12][tx] = __ldg(&A[(I+12)*N + kk*TW+tx]);
		//Bs[ty+12][tx] = __ldg(&B[(kk*TW+ty+12)*N + J]);
		////printf("ty_8\n");
		//As[ty+16][tx] = __ldg(&A[(I+16)*N + kk*TW+tx]);
		//Bs[ty+16][tx] = __ldg(&B[(kk*TW+ty+16)*N + J]);
		//As[ty+20][tx] = __ldg(&A[(I+20)*N + kk*TW+tx]);
		//Bs[ty+20][tx] = __ldg(&B[(kk*TW+ty+20)*N + J]);
		////printf("ty_16\n");
		//As[ty+24][tx] = __ldg(&A[(I+24)*N + kk*TW+tx]);
		//Bs[ty+24][tx] = __ldg(&B[(kk*TW+ty+24)*N + J]);
		//As[ty+28][tx] = __ldg(&A[(I+28)*N + kk*TW+tx]);
		//Bs[ty+28][tx] = __ldg(&B[(kk*TW+ty+28)*N + J]);
		//printf("ty_24\n");
		__syncthreads();
			//for (int k=0; k<TW && k+kk*TW<N; k++)
			#pragma unroll
			for (int k=0; k<min(TW,N-kk*TW); k++)
			{
				Cij += As[ty][k] * Bs[k][tx];
				//Cij_4 += As[ty+4][k] * Bs[k][tx];
				//Cij_8 += As[ty+8][k] * Bs[k][tx];
				//Cij_12 += As[ty+12][k] * Bs[k][tx];
				//Cij_16 += As[ty+16][k] * Bs[k][tx];
				//Cij_20 += As[ty+20][k] * Bs[k][tx];
				//Cij_24 += As[ty+24][k] * Bs[k][tx];
				//Cij_28 += As[ty+28][k] * Bs[k][tx];
			}
		__syncthreads();
	}
		C[I*N + J] = Cij;
		//C[(I+4)*N + J] = Cij_4;
		//C[(I+8)*N + J] = Cij_8;
		//C[(I+12)*N + J] = Cij_12;
		//C[(I+16)*N + J] = Cij_16;
		//C[(I+20)*N + J] = Cij_20;
		//C[(I+24)*N + J] = Cij_24;
		//C[(I+28)*N + J] = Cij_28;
	}
}
else
{
	int I = by*TW + ty; 
	int J = bx*TW + tx;
//	for (int kk=0; kk<ceilf(float (N)/TW); kk++)
	if((I < N) && (J < N)) { 
	#pragma unroll
	for (int kk=0; kk<N/TW; kk++)
	{
		As[ty][tx] = __ldg(&A[I*N + kk*TW + tx]);
		Bs[ty][tx] = __ldg(&B[(kk*TW+ty)*N + J]);
		As[ty+(TW/8)][tx] = __ldg(&A[(I+(TW/8))*N + kk*TW + tx]);
		Bs[ty+(TW/8)][tx] = __ldg(&B[(kk*TW+ty+(TW/8))*N + J]);
		//printf("As = %f A = %f\n",As[ty][tx],A[I*N+kk*TWx+tx]); 
		//printf("ty = %d\n",ty);
		As[ty+(TW/4)][tx] = __ldg(&A[(I+(TW/4))*N + kk*TW+tx]);
		Bs[ty+(TW/4)][tx] = __ldg(&B[(kk*TW+ty+(TW/4))*N + J]);
		As[ty+(3*TW/8)][tx] = __ldg(&A[(I+(3*TW/8))*N + kk*TW+tx]);
		Bs[ty+(3*TW/8)][tx] = __ldg(&B[(kk*TW+ty+(3*TW/8))*N + J]);
		//printf("ty_8\n");
		As[ty+(TW/2)][tx] = __ldg(&A[(I+(TW/2))*N + kk*TW+tx]);
		Bs[ty+(TW/2)][tx] = __ldg(&B[(kk*TW+ty+(TW/2))*N + J]);
		As[ty+(5*TW/8)][tx] = __ldg(&A[(I+(5*TW/8))*N + kk*TW+tx]);
		Bs[ty+(5*TW/8)][tx] = __ldg(&B[(kk*TW+ty+(5*TW/8))*N + J]);
		//printf("ty_16\n");
		As[ty+(3*TW/4)][tx] = __ldg(&A[(I+(3*TW/4))*N + kk*TW+tx]);
		Bs[ty+(3*TW/4)][tx] = __ldg(&B[(kk*TW+ty+(3*TW/4))*N + J]);
		As[ty+(7*TW/8)][tx] = __ldg(&A[(I+(7*TW/8))*N + kk*TW+tx]);
		Bs[ty+(7*TW/8)][tx] = __ldg(&B[(kk*TW+ty+(7*TW/8))*N + J]);
		__syncthreads();
			//for (int k=0; k<TW && k+kk*TW<N; k++)
			#pragma unroll
			for (int k=0; k<TW; k++)
			{
				Cij += As[ty][k] * Bs[k][tx];
				Cij_4 += As[ty+(TW/8)][k] * Bs[k][tx];
				Cij_8 += As[ty+(TW/4)][k] * Bs[k][tx];
				Cij_12 += As[ty+(3*TW/8)][k] * Bs[k][tx];
				Cij_16 += As[ty+(TW/2)][k] * Bs[k][tx];
				Cij_20 += As[ty+(5*TW/8)][k] * Bs[k][tx];
				Cij_24 += As[ty+(3*TW/4)][k] * Bs[k][tx];
				Cij_28 += As[ty+(7*TW/8)][k] * Bs[k][tx];
			}
		__syncthreads();
		}
		C[I*N + J] = Cij;
		C[(I+(TW/8))*N + J] = Cij_4;
		C[(I+(TW/4))*N + J] = Cij_8;
		C[(I+(3*TW/8))*N + J] = Cij_12;
		C[(I+(TW/2))*N + J] = Cij_16;
		C[(I+(5*TW/8))*N + J] = Cij_20;
		C[(I+(3*TW/4))*N + J] = Cij_24;
		C[(I+(7*TW/8))*N + J] = Cij_28;
		}
	}
}
