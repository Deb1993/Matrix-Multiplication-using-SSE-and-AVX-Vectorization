#include "hip/hip_runtime.h"
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include "utils.h"
#include "types.h"
using namespace std;
#define TW 32
#include <stdio.h>
__global__ void matMul(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {
	__shared__ double As[TW][TW], Bs[TW][TW];
	int ty = threadIdx.y, tx = threadIdx.x;
	int by = blockIdx.y, bx = blockIdx.x;
	double Cij = 0;
	double Cij_04 = 0;
	double Cij_08 = 0;
	double Cij_12 = 0;
	double Cij_16 = 0;
	double Cij_20 = 0;
	double Cij_24 = 0;
	double Cij_28 = 0;
	if(N)//%TW)
	{
		int I = by*TW + ty; 
		//int I = min(N-1,by*TW + ty); 
		//int J= min(N-1,bx*TW + tx);
		int J= bx*TW + tx;
		//	for (int kk=0; kk<ceilf(float (N)/TW); kk++)
		if((I < N) && (J < N)){
			#pragma unroll
		for (int kk=0; kk<(N/TW + int(bool(N%TW))); kk++)
			//for (int kk=0; kk<(N/TW + 1); kk++)
			{
			//printf("bx=%d,by=%d,tx=%d,ty=%d,Reading starting for kk=%d\n",bx,by,tx,ty,kk);
		//		As[ty][tx]    = __ldg(&A[I*N + kk*TW+tx]);
		//		As[ty+4][tx]  = __ldg(&A[(I+4)*N + kk*TW+tx]);
		//		As[ty+8][tx]  = __ldg(&A[(I+8)*N + kk*TW+tx]);
		//		As[ty+12][tx] = __ldg(&A[(I+12)*N + kk*TW+tx]);
		//		As[ty+16][tx] = __ldg(&A[(I+16)*N + kk*TW+tx]);
		//		As[ty+20][tx] = __ldg(&A[(I+20)*N + kk*TW+tx]);
		//		As[ty+24][tx] = __ldg(&A[(I+24)*N + kk*TW+tx]);
		//		As[ty+28][tx] = __ldg(&A[(I+28)*N + kk*TW+tx]);
		//		Bs[ty][tx]    = __ldg(&B[(kk*TW+ty)*N + J]);
		//		Bs[ty+4][tx]  = __ldg(&B[(kk*TW+ty+4)*N + J]);
		//		Bs[ty+8][tx]  = __ldg(&B[(kk*TW+ty+8)*N + J]);
		//		Bs[ty+12][tx] = __ldg(&B[(kk*TW+ty+12)*N + J]);
		//		Bs[ty+16][tx] = __ldg(&B[(kk*TW+ty+16)*N + J]);
		//		Bs[ty+20][tx] = __ldg(&B[(kk*TW+ty+20)*N + J]);
		//		Bs[ty+24][tx] = __ldg(&B[(kk*TW+ty+24)*N + J]);
		//		Bs[ty+28][tx] = __ldg(&B[(kk*TW+ty+28)*N + J]);
				As[ty][tx] = __ldg(&A[I*N + kk*TW+tx]);
				As[ty+4][tx] = (I+4 < N)? __ldg(&A[(I+4)*N + kk*TW+tx]):0;
				As[ty+8][tx] = (I+8 < N)? __ldg(&A[(I+8)*N + kk*TW+tx]):0;
				As[ty+12][tx] = (I+12 < N)? __ldg(&A[(I+12)*N + kk*TW+tx]):0;
				As[ty+16][tx] = (I+16 < N)? __ldg(&A[(I+16)*N + kk*TW+tx]):0;
				As[ty+20][tx] = (I+20 < N)? __ldg(&A[(I+20)*N + kk*TW+tx]):0;
				As[ty+24][tx] = (I+24 < N)? __ldg(&A[(I+24)*N + kk*TW+tx]):0;
				As[ty+28][tx] = (I+28 < N)? __ldg(&A[(I+28)*N + kk*TW+tx]):0;
				Bs[ty][tx] = __ldg(&B[(kk*TW+ty)*N + J]);
				Bs[ty+4][tx] = (kk*TW+ty+4 < N)? __ldg(&B[(kk*TW+ty+4)*N + J]):0;
				Bs[ty+8][tx] = (kk*TW+ty+8 < N)? __ldg(&B[(kk*TW+ty+8)*N + J]):0;
				Bs[ty+12][tx] = (kk*TW+ty+12 < N)? __ldg(&B[(kk*TW+ty+12)*N + J]):0;
				Bs[ty+16][tx] = (kk*TW+ty+16 < N)? __ldg(&B[(kk*TW+ty+16)*N + J]):0;
				Bs[ty+20][tx] = (kk*TW+ty+20 < N)? __ldg(&B[(kk*TW+ty+20)*N + J]):0;
				Bs[ty+24][tx] = (kk*TW+ty+24 < N)? __ldg(&B[(kk*TW+ty+24)*N + J]):0;
				Bs[ty+28][tx] = (kk*TW+ty+28 < N)? __ldg(&B[(kk*TW+ty+28)*N + J]):0;
				__syncthreads();
			//printf("bx=%d,by=%d,tx=%d,ty=%d,Reading completed for kk=%d\n",bx,by,tx,ty,kk);
				//for (int k=0; k<TW && k+kk*TW<N; k++)
			#pragma unroll
				for (int k=0; k<min(TW,N-kk*TW); k++)
				{
					Cij    += As[ty][k] * Bs[k][tx];
					Cij_04 += As[ty+4][k] * Bs[k][tx];
					Cij_08 += As[ty+8][k] * Bs[k][tx];
					Cij_12 += As[ty+12][k] * Bs[k][tx];
					Cij_16 += As[ty+16][k] * Bs[k][tx];
					Cij_20 += As[ty+20][k] * Bs[k][tx];
					Cij_24 += As[ty+24][k] * Bs[k][tx];
					Cij_28 += As[ty+28][k] * Bs[k][tx];
				}
				__syncthreads();
			//printf("bx=%d,by=%d,tx=%d,ty=%d,Computing completed for kk=%d\n",bx,by,tx,ty,kk);
			}
			//printf("bx=%d,by=%d,tx=%d,ty=%d,Store starting \n",bx,by,tx,ty);
			C[I*N + J]      = Cij;
			C[(I+4)*N + J]  = Cij_04;
			C[(I+8)*N + J]  = Cij_08;
			C[(I+12)*N + J] = Cij_12;
			C[(I+16)*N + J] = Cij_16;
			C[(I+20)*N + J] = Cij_20;
			C[(I+24)*N + J] = Cij_24;
			C[(I+28)*N + J] = Cij_28;
		//	if(I+4<N) C[(I+4)*N + J]  = Cij_04;
		//	if(I+8<N) C[(I+8)*N + J]  = Cij_08;
		//	if(I+12<N) C[(I+12)*N + J] = Cij_12;
		//	if(I+16<N) C[(I+16)*N + J] = Cij_16;
		//	if(I+20<N) C[(I+20)*N + J] = Cij_20;
		//	if(I+24<N) C[(I+24)*N + J] = Cij_24;
		//	if(I+28<N) C[(I+28)*N + J] = Cij_28;
			//printf("bx=%d,by=%d,tx=%d,ty=%d,Store completed\n",bx,by,tx,ty);
		}
	}
	else
	{
		int I = by*TW + ty; 
		int J = bx*TW + tx;
		//	for (int kk=0; kk<ceilf(float (N)/TW); kk++)

		if((I < N) && (J < N)){
			#pragma unroll
			for (int kk=0; kk<N/TW; kk++)
			{
				As[ty][tx]    = __ldg(&A[I*N + kk*TW+tx]);
				As[ty+4][tx]  = __ldg(&A[(I+4)*N + kk*TW+tx]);
				As[ty+8][tx]  = __ldg(&A[(I+8)*N + kk*TW+tx]);
				As[ty+12][tx] = __ldg(&A[(I+12)*N + kk*TW+tx]);
				As[ty+16][tx] = __ldg(&A[(I+16)*N + kk*TW+tx]);
				As[ty+20][tx] = __ldg(&A[(I+20)*N + kk*TW+tx]);
				As[ty+24][tx] = __ldg(&A[(I+24)*N + kk*TW+tx]);
				As[ty+28][tx] = __ldg(&A[(I+28)*N + kk*TW+tx]);
				Bs[ty][tx]    = __ldg(&B[(kk*TW+ty)*N + J]);
				Bs[ty+4][tx]  = __ldg(&B[(kk*TW+ty+4)*N + J]);
				Bs[ty+8][tx]  = __ldg(&B[(kk*TW+ty+8)*N + J]);
				Bs[ty+12][tx] = __ldg(&B[(kk*TW+ty+12)*N + J]);
				Bs[ty+16][tx] = __ldg(&B[(kk*TW+ty+16)*N + J]);
				Bs[ty+20][tx] = __ldg(&B[(kk*TW+ty+20)*N + J]);
				Bs[ty+24][tx] = __ldg(&B[(kk*TW+ty+24)*N + J]);
				Bs[ty+28][tx] = __ldg(&B[(kk*TW+ty+28)*N + J]);
				__syncthreads();
				//for (int k=0; k<TW && k+kk*TW<N; k++)
			#pragma unroll
				for (int k=0; k<TW; k++)
				{
					Cij    += As[ty][k] * Bs[k][tx];
					Cij_04 += As[ty+4][k] * Bs[k][tx];
					Cij_08 += As[ty+8][k] * Bs[k][tx];
					Cij_12 += As[ty+12][k] * Bs[k][tx];
					Cij_16 += As[ty+16][k] * Bs[k][tx];
					Cij_20 += As[ty+20][k] * Bs[k][tx];
					Cij_24 += As[ty+24][k] * Bs[k][tx];
					Cij_28 += As[ty+28][k] * Bs[k][tx];
				}
				__syncthreads();
			}
			C[I*N + J]      = Cij;
			C[(I+4)*N + J]  = Cij_04;
			C[(I+8)*N + J]  = Cij_08;
			C[(I+12)*N + J] = Cij_12;
			C[(I+16)*N + J] = Cij_16;
			C[(I+20)*N + J] = Cij_20;
			C[(I+24)*N + J] = Cij_24;
			C[(I+28)*N + J] = Cij_28;
		}
	}
}
