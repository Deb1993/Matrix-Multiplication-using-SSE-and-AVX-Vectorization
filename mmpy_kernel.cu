#include "hip/hip_runtime.h"
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include "utils.h"
#include "types.h"
using namespace std;
#define TW 32
#include <stdio.h>
__global__ void matMul(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {
	__shared__ double As[TW][TW], Bs[TW][TW];
	int ty = threadIdx.y, tx = threadIdx.x;
	int by = blockIdx.y, bx = blockIdx.x;
	double Cij = 0;
	if(N%TW)
{
	int I = min(N-1,by*TW + ty); 
	int J= min(N-1,bx*TW + tx);
//	for (int kk=0; kk<ceilf(float (N)/TW); kk++)
	for (int kk=0; kk<(N/TW + int(bool(N%TW))); kk++)
	{
		As[ty][tx] = A[I*N + kk*TW+tx];
		Bs[ty][tx] = B[(kk*TW+ty)*N + J];
		__syncthreads();
			//for (int k=0; k<TW && k+kk*TW<N; k++)
			for (int k=0; k<min(TW,N-kk*TW); k++)
			{
				Cij+= As[ty][k] * Bs[k][tx];
			}
		__syncthreads();
	}
		C[I*N + J] = Cij;
}
else
{
	int I = by*TW + ty; 
	int J= bx*TW + tx;
//	for (int kk=0; kk<ceilf(float (N)/TW); kk++)
	for (int kk=0; kk<N/TW; kk++)
	{
		As[ty][tx] = A[I*N + kk*TW+tx];
		Bs[ty][tx] = B[(kk*TW+ty)*N + J];
		__syncthreads();
			//for (int k=0; k<TW && k+kk*TW<N; k++)
			for (int k=0; k<TW; k++)
			{
				Cij+= As[ty][k] * Bs[k][tx];
			}
		__syncthreads();
		}
		C[I*N + J] = Cij;

	}
}
