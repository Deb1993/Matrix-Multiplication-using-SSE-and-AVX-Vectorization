#include "hip/hip_runtime.h"
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include <stdio.h>
#include "utils.h"
#include "types.h"
using namespace std;

__global__ void matMul(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {

//    int I =  blockIdx.y*blockDim.y + threadIdx.y;
//    int J =  blockIdx.x*blockDim.x + threadIdx.x;
//
//    if((I < N) && (J < N)){
//        _DOUBLE_ _c = 0;
//        for (unsigned int k = 0; k < N; k++) {
//            _DOUBLE_ a = A[I * N + k];
//            _DOUBLE_ b = B[k * N + J];
//            _c += a * b;
//        }
//        C[I * N + J] = _c;
//    }
#define TW 32 

__shared__ double AS[TW][TW], BS[TW][TW];

int ty = threadIdx.y;
int tx = threadIdx.x;
int bx = blockIdx.x;
int by = blockIdx.y;

int I = by*TW + ty;
int J = bx*TW + tx;
double Cij = 0;

if( (I < N) && (J < N)) {
	for (int kk = 0 ; kk < N/TW ; kk++) {
		//printf("Inside kk loop = %d\n",kk);
		//printf("Inside kk loop = %d\n",kk);
		AS[ty][tx] = A[I*N + kk*TW + tx];
		BS[ty][tx] = B[(kk*TW + ty)*N  + J];
		__syncthreads();
		for(int k = 0; k < TW ; k++) {
			Cij += AS[ty][k] * BS[k][tx];
		//printf("Inside k loop = %d\n",k);
		 }
		__syncthreads();
		}
	C[I*N + J] = Cij;
	}
}
