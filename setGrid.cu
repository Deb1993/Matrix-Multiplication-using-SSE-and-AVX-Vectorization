
#include <hip/hip_runtime.h>
void setGrid(int n, dim3 &blockDim, dim3 &gridDim)
{
   // set your block dimensions and grid dimensions here
   gridDim.x = n / blockDim.x;
   gridDim.y = n / blockDim.y;
   //gridDim.x = 24;
   //gridDim.y = 16;
   if(n % blockDim.x != 0)
   	gridDim.x++;
   if(n % blockDim.y != 0)
    	gridDim.y++;
}
